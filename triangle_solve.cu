#include "hip/hip_runtime.h"
#include "triangle_solve.h"

template<typename Float>
SparseTriangularSolver<Float>::SparseTriangularSolver(uint n_rows, uint n_entries, uint* rows, uint* cols, Float* data, bool lower) : m_lower(lower), m_n_rows(n_rows) {
	std::vector<uint> levels;//TODO: this could be an array, as it always has as many elements as rows
	levels.reserve(n_rows);

	m_level_ptr_h.reserve(n_rows + 1); // Worst-case size (100% dense triangular matrix)
	m_level_ptr_h.push_back(0);

	m_chain_ptr.reserve(n_rows + 1); // TODO: could use a more educated guess depending on block size and n_rows
	m_chain_ptr.push_back(0);

	Float* dag_values = (Float*) malloc(n_entries*sizeof(Float));
	std::copy(data, data+n_entries, dag_values);

	std::vector<uint> candidates;
	for(uint i = 0; i < n_rows; i++)
		candidates.push_back(i);

	uint level = 0;
	uint level_idx = 0;
	while(levels.size() < n_rows) {
		uint level_size = 0;
		for(uint candidate : candidates) {
			bool independent = true;
			for(uint j=rows[candidate]; j<rows[candidate + 1]; j++) {
				if (cols[j] != candidate && dag_values[j] != 0.0f) {
					independent = false;
					break;
				}
			}

			if (independent) {
				levels.push_back(candidate);
				level_size++;
			}
		}

		// Sort indices in the current level
		std::sort(levels.end() - level_size, levels.end());

		candidates.clear();
		for (int i=level; i<level+level_size; i++) {
			int row = levels[i];
			for (int j=0; j<n_rows; j++) {
				for (int k=rows[j]; k<rows[j+1]; k++) {
					if (cols[k] == row && cols[k] != j) {
						dag_values[k] = 0.0f;

						bool candidate = true;
						// TODO: replace this with a set
						for (uint c : candidates) {
							if (c == j) {
								candidate = false;
								break;
							}
						}
						if (candidate)
							candidates.push_back(j);
					}
				}
			}
		}

		level += level_size;
		level_idx++;
		if (level_size > BLOCK_SIZE) {
			if (m_chain_ptr.back() == level_idx-1)
				m_chain_ptr.push_back(level_idx);
			else {
				m_chain_ptr.push_back(level_idx-1);
				m_chain_ptr.push_back(level_idx);
			}
		}
		m_level_ptr_h.push_back(level);
	}
	if (m_chain_ptr.back() != level_idx)
		m_chain_ptr.push_back(level_idx);

	m_chain_ptr.shrink_to_fit();
	m_level_ptr_h.shrink_to_fit();

	free(dag_values);

	// Copy stuff to the GPU
	// CSR Matrix arrays
    hipMalloc((void **)&m_rows_d, (1+n_rows)*sizeof(uint));
    hipMemcpy(m_rows_d, rows, (1+n_rows)*sizeof(uint), hipMemcpyHostToDevice);
    hipMalloc((void **)&m_cols_d, n_entries*sizeof(uint));
    hipMemcpy(m_cols_d, cols, n_entries*sizeof(uint), hipMemcpyHostToDevice);
    hipMalloc((void **)&m_data_d, n_entries*sizeof(Float));
    hipMemcpy(m_data_d, data, n_entries*sizeof(Float), hipMemcpyHostToDevice);
	// Solve data structure
    hipMalloc((void **)&m_level_ptr_d, m_level_ptr_h.size()*sizeof(uint));
    hipMemcpy(m_level_ptr_d, &m_level_ptr_h[0], m_level_ptr_h.size()*sizeof(uint), hipMemcpyHostToDevice);
    hipMalloc((void **)&m_levels_d, n_rows*sizeof(uint));
    hipMemcpy(m_levels_d, &levels[0], n_rows*sizeof(uint), hipMemcpyHostToDevice);
	// RHS and solution
	hipMalloc((void**)&m_b_d, n_rows*sizeof(Float));
	hipMalloc((void**)&m_x_d, n_rows*sizeof(Float));
}

template<typename Float>
SparseTriangularSolver<Float>::~SparseTriangularSolver() {
	hipFree(m_rows_d);
	hipFree(m_cols_d);
	hipFree(m_data_d);
	hipFree(m_level_ptr_d);
	hipFree(m_levels_d);
	hipFree(m_b_d);
	hipFree(m_x_d);
}

template<typename Float>
Float* SparseTriangularSolver<Float>::solve(Float *b) {

	hipMemcpy(m_b_d, b, m_n_rows*sizeof(Float), hipMemcpyHostToDevice);
	hipMemcpy(m_x_d, b, m_n_rows*sizeof(Float), hipMemcpyHostToDevice);//TODO: device to device copy instead?

	for (int i=0; i<m_chain_ptr.size()-1; i++) {
        if (m_chain_ptr[i]+1 == m_chain_ptr[i+1]){
			// Multi block kernel
			//TODO: this requires storing level_ptr on the CPU, is this really necessary?
			int num_blocks = (m_level_ptr_h[m_chain_ptr[i+1]] - m_level_ptr_h[m_chain_ptr[i]] + BLOCK_SIZE - 1) / BLOCK_SIZE;
			solve_row_multiblock<<<num_blocks, BLOCK_SIZE>>>(m_chain_ptr[i], m_level_ptr_d, m_levels_d, m_rows_d, m_cols_d, m_data_d, m_b_d, m_x_d, m_lower);
		} else {
			// Chain fits in one block
			solve_chain<<<1, BLOCK_SIZE>>>(m_chain_ptr[i], m_chain_ptr[i+1], m_level_ptr_d, m_levels_d, m_rows_d, m_cols_d, m_data_d, m_b_d, m_x_d, m_lower);
		}
        hipDeviceSynchronize();
	}

	Float *x_h = (Float*) malloc(m_n_rows*sizeof(Float));
	hipMemcpy(x_h, m_x_d, m_n_rows*sizeof(Float), hipMemcpyDeviceToHost);
	return x_h;
}

template<typename Float>
__global__ void solve_row_multiblock(uint level, uint* level_ptr, uint *levels, uint* rows, uint* columns, Float* values, Float* b, Float* x, bool lower) {
    uint row_idx = level_ptr[level] + blockDim.x * blockIdx.x + threadIdx.x;
	if (row_idx >= level_ptr[level+1])
		return;
	uint row = levels[row_idx];
	uint row_start = rows[row];
	uint row_end = rows[row + 1];
	uint diag_ptr;
	if (lower) {
		diag_ptr = row_end - 1;
		row_end--;
	} else {
		diag_ptr = row_start;
		row_start++;
	}

	Float r = 0.0f;
	for (uint i=row_start; i<row_end; i++) {
		r += values[i]*x[columns[i]];
	}

	x[row] -= r;
	x[row] /= values[diag_ptr];
}

template<typename Float>
__global__ void solve_chain(uint chain_start, uint chain_end, uint *level_ptr, uint *levels, uint* rows, uint* columns, Float* values, Float* b, Float* x, bool lower) {

	for (uint level=chain_start; level<chain_end; level++) {
		uint level_start = level_ptr[level];
		uint level_end = level_ptr[level+1];
		uint row_idx = level_start + threadIdx.x;
		if (row_idx >= level_end)
			continue;
		uint row = levels[row_idx];
		uint row_start = rows[row];
		uint row_end = rows[row + 1];
		uint diag_ptr;
		if (lower) {
			diag_ptr = row_end - 1;
			row_end--;
		} else {
			diag_ptr = row_start;
			row_start++;
		}

		Float r = 0.0f;
		for (uint i=row_start; i<row_end; i++) {
			r += values[i]*x[columns[i]];
		}

		x[row] -= r;
		x[row] /= values[diag_ptr];
		__syncthreads(); // Synchronize before moving to next level
	}
}

int main(void) {

	typedef float Float;

    uint n_rows = 9;
    uint n_entries = 18;

	uint rows_h[] = {0, 1, 2, 3, 5, 7, 9, 11, 14, 18};
	uint columns_h[] = {0, 1, 2, 0, 3, 0, 4, 1, 5, 2, 6, 3, 4, 7, 2, 3, 4, 8};
	Float values_h[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f};


	SparseTriangularSolver<Float> solver(n_rows, n_entries, rows_h, columns_h, values_h, true);

	Float b_h[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};

	Float *x_h = solver.solve(b_h);

	for (int i=0; i<9; i++)
		std::cout << x_h[i] << " ";
	std::cout << std::endl;

	return 0;
}

